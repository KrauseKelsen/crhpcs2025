#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "matrix.h"

__global__ void transpose_matrix_naive_gpu(int n, int m, const float *origin, float *result)
{
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < n && j < m)
  {
    result[j * m + i] = origin[i * n +j];
  }
}

void run_transpose_naive_gpu(const int n, const int m, const float *host_origin)
{
  float *dev_origin, *dev_result;
  hipMalloc(&dev_origin, n * m * sizeof(float));
  hipMalloc(&dev_result, n * m * sizeof(float));

  hipMemcpy(dev_origin, host_origin, M * N * sizeof(float), hipMemcpyHostToDevice);
}

int main(int argc, char **argv)
{
  printf("Hello, world!\n");
  int n = atol(argv[1]);
  int m = atol(argv[2]);
  float **matA = alloc_matrix(n, m);
  float **matB = alloc_matrix(m, n);
  init_matrix(n, m, 2, matA);
  //transpose_matrix_block(n, m, matA, matB);
  free_matrix(n, m, matA);
  free_matrix(m, n, matB);
  return 0;
}
